#include "hip/hip_runtime.h"
#include "pathtrace.h"

#include <cstdio>
#include <hip/hip_runtime.h>
#include <cmath>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <thrust/partition.h>

#include "sceneStructs.h"
#include "scene.h"
#include "glm/glm.hpp"
#include "glm/gtx/norm.hpp"
#include "utilities.h"
#include "intersections.h"
#include "interactions.h"
#include "OpenImageDenoise/oidn.hpp"

#define ERRORCHECK 1
#define ANTIALIAS 0
#define DOF 0
#define DENOISE 1
#define DENOISE_INTERVAL 50

#define FILENAME (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define checkCUDAError(msg) checkCUDAErrorFn(msg, FILENAME, __LINE__)
void checkCUDAErrorFn(const char* msg, const char* file, int line)
{
#if ERRORCHECK
    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    if (hipSuccess == err)
    {
        return;
    }

    fprintf(stderr, "CUDA error");
    if (file)
    {
        fprintf(stderr, " (%s:%d)", file, line);
    }
    fprintf(stderr, ": %s: %s\n", msg, hipGetErrorString(err));
#ifdef _WIN32
    getchar();
#endif // _WIN32
    exit(EXIT_FAILURE);
#endif // ERRORCHECK
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth)
{
    int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
    return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* pbo, glm::ivec2 resolution, int iter, glm::vec3* image)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < resolution.x && y < resolution.y)
    {
        int index = x + (y * resolution.x);
        glm::vec3 pix = image[index];

        glm::ivec3 color;
        color.x = glm::clamp((int)(pix.x / iter * 255.0), 0, 255);
        color.y = glm::clamp((int)(pix.y / iter * 255.0), 0, 255);
        color.z = glm::clamp((int)(pix.z / iter * 255.0), 0, 255);

        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

static Scene* hst_scene = NULL;
static GuiDataContainer* guiData = NULL;
static glm::vec3* dev_image = NULL;
static Geom* dev_geoms = NULL;
static Material* dev_materials = NULL;
static PathSegment* dev_paths = NULL;
static ShadeableIntersection* dev_intersections = NULL;
static Vertex* dev_vertices = NULL;
static Texture* dev_textures = NULL;
static glm::vec4* dev_pixels = NULL;
static glm::vec3* dev_output = NULL;
static glm::vec3* dev_albedo = NULL;
static glm::vec3* dev_normal = NULL;

// TODO: static variables for device memory, any extra info you need, etc
// ...

void InitDataContainer(GuiDataContainer* imGuiData)
{
    guiData = imGuiData;
}

void pathtraceInit(Scene* scene)
{
    hst_scene = scene;

    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    hipMalloc(&dev_image, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_image, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_paths, pixelcount * sizeof(PathSegment));

    hipMalloc(&dev_geoms, scene->geoms.size() * sizeof(Geom));
    hipMemcpy(dev_geoms, scene->geoms.data(), scene->geoms.size() * sizeof(Geom), hipMemcpyHostToDevice);

    hipMalloc(&dev_materials, scene->materials.size() * sizeof(Material));
    hipMemcpy(dev_materials, scene->materials.data(), scene->materials.size() * sizeof(Material), hipMemcpyHostToDevice);

    hipMalloc(&dev_intersections, pixelcount * sizeof(ShadeableIntersection));
    hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

    hipMalloc(&dev_vertices, scene->vertices.size() * sizeof(Vertex));
    hipMemcpy(dev_vertices, scene->vertices.data(), scene->vertices.size() * sizeof(Vertex), hipMemcpyHostToDevice);

    hipMalloc(&dev_textures, scene->textures.size() * sizeof(Texture));
    hipMemcpy(dev_textures, scene->textures.data(), scene->textures.size() * sizeof(Texture), hipMemcpyHostToDevice);

    hipMalloc(&dev_pixels, scene->texturePixels.size() * sizeof(glm::vec4));
    hipMemcpy(dev_pixels, scene->texturePixels.data(), scene->texturePixels.size() * sizeof(glm::vec4), hipMemcpyHostToDevice);

    hipMalloc(&dev_output, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_output, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_albedo, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_albedo, 0, pixelcount * sizeof(glm::vec3));

    hipMalloc(&dev_normal, pixelcount * sizeof(glm::vec3));
    hipMemset(dev_normal, 0, pixelcount * sizeof(glm::vec3));

    checkCUDAError("pathtraceInit");
}

void pathtraceFree()
{
    hipFree(dev_image);  // no-op if dev_image is null
    hipFree(dev_paths);
    hipFree(dev_geoms);
    hipFree(dev_materials);
    hipFree(dev_intersections);
    hipFree(dev_vertices);
    hipFree(dev_textures);
    hipFree(dev_pixels);
    hipFree(dev_output);
    hipFree(dev_albedo);
    hipFree(dev_normal);

    checkCUDAError("pathtraceFree");
}

/**
* Generate PathSegments with rays from the camera through the screen into the
* scene, which is the first bounce of rays.
*
* Antialiasing - add rays for sub-pixel sampling
* motion blur - jitter rays "in time"
* lens effect - jitter ray origin positions based on a lens
*/
__global__ void generateRayFromCamera(Camera cam, int iter, int traceDepth, PathSegment* pathSegments)
{
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < cam.resolution.x && y < cam.resolution.y) {
        int index = x + (y * cam.resolution.x);
        PathSegment& segment = pathSegments[index];

        segment.ray.origin = cam.position;
        segment.color = glm::vec3(1.0f, 1.0f, 1.0f);

        thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, pathSegments[index].remainingBounces);

        float x_rng = 0.0f;
        float y_rng = 0.0f;

        if (ANTIALIAS)
        {
            // implement antialiasing by jittering the ray
            thrust::uniform_real_distribution<float> u05(-0.5, 0.5);

            x_rng = u05(rng);
            y_rng = u05(rng);
        }

        segment.ray.direction = glm::normalize(cam.view
            - cam.right * cam.pixelLength.x * ((float)x + x_rng - (float)cam.resolution.x * 0.5f)
            - cam.up * cam.pixelLength.y * ((float)y + y_rng - (float)cam.resolution.y * 0.5f)
        );
        
        if (DOF)
        {
            thrust::uniform_real_distribution<float> u01(0, 1);

            float angle = u01(rng) * TWO_PI;
            float r = u01(rng) * cam.lens_radius;
            glm::vec3 offset = r * (cos(angle) * cam.right + sin(angle) * cam.up);

            segment.ray.origin = cam.position + offset;
            segment.ray.direction = glm::normalize(cam.focal_dis / glm::dot(segment.ray.direction, cam.view) * segment.ray.direction - offset);
        }

        segment.pixelIndex = index;
        segment.remainingBounces = traceDepth;
    }
}

// M�ller�Trumbore intersection
__host__ __device__ float triangleIntersect(glm::vec3 p0, glm::vec3 p1, glm::vec3 p2,
    glm::vec3 rayOrigin, glm::vec3 rayDirection) {
    glm::vec3 edge1, edge2, h, s, q;
    float a, f, u, v;
    edge1 = p1 - p0;
    edge2 = p2 - p0;
    h = glm::cross(rayDirection, edge2);
    a = glm::dot(edge1, h);
    if (a > -EPSILON && a < EPSILON) {
        return INFINITY;    // This ray is parallel to this triangle.
    }
    f = 1.0 / a;
    s = rayOrigin - p0;
    u = f * glm::dot(s, h);
    if (u < 0.0 || u > 1.0)
        return INFINITY;
    q = glm::cross(s, edge1);
    v = f * glm::dot(rayDirection, q);
    if (v < 0.0 || u + v > 1.0) {
        return INFINITY;
    }
    // At this stage we can compute t to find out where the intersection point is on the line.
    float t = f * glm::dot(edge2, q);
    if (t > EPSILON) {
        return t;
    }
    else // This means that there is a line intersection but not a ray intersection.
        return INFINITY;
}

__host__ __device__ glm::vec3 barycentric(glm::vec3 p, glm::vec3 t1, glm::vec3 t2, glm::vec3 t3) {
    glm::vec3 edge1 = t2 - t1;
    glm::vec3 edge2 = t3 - t2;
    float S = length(cross(edge1, edge2));

    edge1 = p - t2;
    edge2 = p - t3;
    float S1 = length(cross(edge1, edge2));

    edge1 = p - t1;
    edge2 = p - t3;
    float S2 = length(cross(edge1, edge2));

    edge1 = p - t1;
    edge2 = p - t2;
    float S3 = length(cross(edge1, edge2));

    return glm::vec3(S1 / S, S2 / S, S3 / S);
}

// computeIntersections handles generating ray intersections ONLY.
// Generating new rays is handled in your shader(s).
// Feel free to modify the code below.
__global__ void computeIntersections(
    int depth,
    int num_paths,
    PathSegment* pathSegments,
    Geom* geoms,
    int geoms_size,
    ShadeableIntersection* intersections,
    Vertex* vertices,
    int vert_size)
{
    int path_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (path_index < num_paths)
    {
        PathSegment pathSegment = pathSegments[path_index];

        float t;
        glm::vec3 intersect_point;
        glm::vec3 normal;
        glm::vec2 uv;
        float t_min = FLT_MAX;
        int mat_index = -1;
        bool outside = true;

        glm::vec3 tmp_intersect;
        glm::vec3 tmp_normal;
        glm::vec3 tmp_tangent;

        // naive parse through global geoms

        for (int i = 0; i < geoms_size; i++)
        {
            Geom& geom = geoms[i];

            if (geom.type == CUBE)
            {
                t = boxIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            else if (geom.type == SPHERE)
            {
                t = sphereIntersectionTest(geom, pathSegment.ray, tmp_intersect, tmp_normal, outside);
            }
            // TODO: add more intersection tests here... triangle? metaball? CSG?

            // Compute the minimum t from the intersection tests to determine what
            // scene geometry object was hit first.
            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                mat_index = geoms[i].materialid;
                normal = tmp_normal;
            }
        }

        // check if intersects with model
        for (int vertIdx = 0; vertIdx < vert_size; vertIdx += 3)
        {
            Vertex& v1 = vertices[vertIdx];
            Vertex& v2 = vertices[vertIdx + 1];
            Vertex& v3 = vertices[vertIdx + 2];
            t = triangleIntersect(v1.pos, v2.pos, v3.pos, pathSegment.ray.origin, pathSegment.ray.direction);

            if (t > 0.0f && t_min > t)
            {
                t_min = t;
                mat_index = v1.materialid;
                glm::vec3 p = pathSegment.ray.origin + (pathSegment.ray.direction * t);
                glm::vec3 bary = barycentric(p, v1.pos, v2.pos, v3.pos); // Calculate barycentric coordinates
                normal = glm::normalize(bary.x * v1.nor + bary.y * v2.nor + bary.z * v3.nor); // Interpolate normals
                uv = bary.x * v1.uv + bary.y * v2.uv + bary.z * v3.uv; // Interpolate uv
                tmp_tangent = bary.x * v1.tangent + bary.y * v2.tangent + bary.z * v3.tangent;
            }
        }

        if (mat_index == -1)
        {
            intersections[path_index].t = -1.0f;
        }
        else
        {
            // The ray hits something
            intersections[path_index].t = t_min;
            intersections[path_index].materialId = mat_index;
            intersections[path_index].surfaceNormal = normal;
            intersections[path_index].surfaceUV = uv;
            intersections[path_index].tangent = tmp_tangent;
        }
    }
}

// LOOK: "fake" shader demonstrating what you might do with the info in
// a ShadeableIntersection, as well as how to use thrust's random number
// generator. Observe that since the thrust random number generator basically
// adds "noise" to the iteration, the image should start off noisy and get
// cleaner as more iterations are computed.
//
// Note that this shader does NOT do a BSDF evaluation!
// Your shaders should handle that - this can allow techniques such as
// bump mapping.
__global__ void shadeFakeMaterial(
    int iter,
    int num_paths,
    ShadeableIntersection* shadeableIntersections,
    PathSegment* pathSegments,
    Material* materials,
    Texture* textures,
    glm::vec4* pixels,
    glm::vec3* dev_albedo,
    glm::vec3* dev_normal)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_paths)
    {
        if (pathSegments[idx].remainingBounces == 0) return;

        ShadeableIntersection intersection = shadeableIntersections[idx];
        if (intersection.t > 0.0f) // if the intersection exists...
        {
          // Set up the RNG
          // LOOK: this is how you use thrust's RNG! Please look at
          // makeSeededRandomEngine as well.
            thrust::default_random_engine rng = makeSeededRandomEngine(iter, idx, pathSegments[idx].remainingBounces);
            thrust::uniform_real_distribution<float> u01(0, 1);

            Material material = materials[intersection.materialId];
            glm::vec3 materialColor = material.color;

            // overwrite color if using material texture
            if (material.diffuse_textureId != -1)
            {
                Texture t = textures[material.diffuse_textureId];
                glm::vec2 uv = intersection.surfaceUV;
                int x = (int)(glm::fract(uv.x) * (float)t.width);
                int y = (int)(glm::fract(1.0 - uv.y) * (float)t.width);

                int pixIdx = t.startIdx + y * t.width + x;

                materialColor = glm::vec3(pixels[pixIdx]);
            }

            glm::vec3 normal = intersection.surfaceNormal;

            // overwrite normal if using normal texture
            if (material.normal_textureId != -1)
            {
                Texture t = textures[material.normal_textureId];
                glm::vec2 uv = intersection.surfaceUV;
                int x = (int)(glm::fract(uv.x) * (float)t.width);
                int y = (int)(glm::fract(1.0 - uv.y) * (float)t.width);

                int pixIdx = t.startIdx + y * t.width + x;

                glm::vec3 pix_nor = glm::vec3(pixels[pixIdx]);

                glm::vec3 tangent = intersection.tangent;
                glm::vec3 bitangent = glm::cross(normal, tangent);

                glm::mat3 mat{ tangent, bitangent, normal };

                normal = mat * pix_nor;
            }

            material.color = materialColor;

            // If the material indicates that the object was a light, "light" the ray
            if (material.emittance > 0.0f) {
                // Light source
                pathSegments[idx].color *= (materialColor * material.emittance);
                pathSegments[idx].remainingBounces = 0;  // Terminate path

#if DENOISE 
                // calculate the albedo and normal data and store them directly
                dev_albedo[pathSegments[idx].pixelIndex] = pathSegments[idx].color;
                dev_normal[pathSegments[idx].pixelIndex] = intersection.surfaceNormal;
#endif
            }
            // Otherwise, do some pseudo-lighting computation. This is actually more
            // like what you would expect from shading in a rasterizer like OpenGL.
            else {
                Ray ray = pathSegments[idx].ray;
                scatterRay(pathSegments[idx], ray.origin + ray.direction * intersection.t, normal, material, rng);

            }

            // If there was no intersection, color the ray black.
            // Lots of renderers use 4 channel color, RGBA, where A = alpha, often
            // used for opacity, in which case they can indicate "no opacity".
            // This can be useful for post-processing and image compositing.
        }
        else {
            pathSegments[idx].color = glm::vec3(0.0f);
        }
    }
}

__global__ void blendImage(glm::vec3* image, glm::vec3* output, int pixelCount)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < pixelCount) {
        image[idx] = image[idx] * (1 - 0.5f) + output[idx] * 0.5f;
    }
}

void denoiseImage() {
    const int width = hst_scene->state.camera.resolution.x;
    const int height = hst_scene->state.camera.resolution.y;
    // Create an Open Image Denoise device
    oidn::DeviceRef device = oidn::newDevice();
    device.commit();
    // Create a filter for denoising a beauty (color) image using optional auxiliary images too
    oidn::FilterRef filter = device.newFilter("RT");  // generic ray tracing filter
    filter.setImage("color", dev_image, oidn::Format::Float3, width, height); // beauty
    filter.setImage("normal", dev_normal, oidn::Format::Float3, width, height);
    filter.setImage("albedo", dev_albedo, oidn::Format::Float3, width, height);
    filter.setImage("output", dev_output, oidn::Format::Float3, width, height);
    filter.set("hdr", true); // beauty image is HDR
    filter.set("cleanAux", true); // auxiliary images will be prefiltered
    filter.commit();
    // Create separate filters for denoising auxiliary albedo & normal image (in-place)
    oidn::FilterRef albedoFilter = device.newFilter("RT"); // same filter type as for beauty
    albedoFilter.setImage("albedo", dev_albedo, oidn::Format::Float3, width, height);
    albedoFilter.setImage("output", dev_albedo, oidn::Format::Float3, width, height);
    albedoFilter.commit();
    oidn::FilterRef normalFilter = device.newFilter("RT");
    normalFilter.setImage("normal", dev_normal, oidn::Format::Float3, width, height);
    normalFilter.setImage("output", dev_normal, oidn::Format::Float3, width, height);
    normalFilter.commit();
    // Prefilter the auxiliary images
    albedoFilter.execute();
    normalFilter.execute();
    // Filter the beauty image
    filter.execute();
    // Check for errors
    const char* errorMessage;
    if (device.getError(errorMessage) != oidn::Error::None) {
        std::cerr << "Error! " << errorMessage << std::endl;
    }
}

// Add the current iteration's output to the overall image
__global__ void finalGather(int nPaths, glm::vec3* image, PathSegment* iterationPaths)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (index < nPaths)
    {
        PathSegment iterationPath = iterationPaths[index];
        image[iterationPath.pixelIndex] += iterationPath.color;
    }
}

/**
 * Wrapper for the __global__ call that sets up the kernel calls and does a ton
 * of memory management
 */
void pathtrace(uchar4* pbo, int frame, int iter)
{
    const int traceDepth = hst_scene->state.traceDepth;
    const Camera& cam = hst_scene->state.camera;
    const int pixelcount = cam.resolution.x * cam.resolution.y;

    // 2D block for generating ray from camera
    const dim3 blockSize2d(8, 8);
    const dim3 blocksPerGrid2d(
        (cam.resolution.x + blockSize2d.x - 1) / blockSize2d.x,
        (cam.resolution.y + blockSize2d.y - 1) / blockSize2d.y);

    // 1D block for path tracing
    const int blockSize1d = 128;

    ///////////////////////////////////////////////////////////////////////////

    // Recap:
    // * Initialize array of path rays (using rays that come out of the camera)
    //   * You can pass the Camera object to that kernel.
    //   * Each path ray must carry at minimum a (ray, color) pair,
    //   * where color starts as the multiplicative identity, white = (1, 1, 1).
    //   * This has already been done for you.
    // * For each depth:
    //   * Compute an intersection in the scene for each path ray.
    //     A very naive version of this has been implemented for you, but feel
    //     free to add more primitives and/or a better algorithm.
    //     Currently, intersection distance is recorded as a parametric distance,
    //     t, or a "distance along the ray." t = -1.0 indicates no intersection.
    //     * Color is attenuated (multiplied) by reflections off of any object
    //   * TODO: Stream compact away all of the terminated paths.
    //     You may use either your implementation or `thrust::remove_if` or its
    //     cousins.
    //     * Note that you can't really use a 2D kernel launch any more - switch
    //       to 1D.
    //   * TODO: Shade the rays that intersected something or didn't bottom out.
    //     That is, color the ray by performing a color computation according
    //     to the shader, then generate a new ray to continue the ray path.
    //     We recommend just updating the ray's PathSegment in place.
    //     Note that this step may come before or after stream compaction,
    //     since some shaders you write may also cause a path to terminate.
    // * Finally, add this iteration's results to the image. This has been done
    //   for you.

    // TODO: perform one iteration of path tracing

    generateRayFromCamera<<<blocksPerGrid2d, blockSize2d>>>(cam, iter, traceDepth, dev_paths);
    checkCUDAError("generate camera ray");

    int depth = 0;
    PathSegment* dev_path_end = dev_paths + pixelcount;
    int num_paths = dev_path_end - dev_paths;

    // --- PathSegment Tracing Stage ---
    // Shoot ray into scene, bounce between objects, push shading chunks

    bool iterationComplete = false;
    while (!iterationComplete)
    {
        // clean shading chunks
        hipMemset(dev_intersections, 0, pixelcount * sizeof(ShadeableIntersection));

        // tracing
        dim3 numblocksPathSegmentTracing = (num_paths + blockSize1d - 1) / blockSize1d;
        computeIntersections<<<numblocksPathSegmentTracing, blockSize1d>>> (
            depth,
            num_paths,
            dev_paths,
            dev_geoms,
            hst_scene->geoms.size(),
            dev_intersections,
            dev_vertices,
            (int)hst_scene->vertices.size()
        );
        checkCUDAError("trace one bounce");
        hipDeviceSynchronize();
        depth++;

        // --- Shading Stage ---
        // Shade path segments based on intersections and generate new rays by
        // evaluating the BSDF.
        // Start off with just a big kernel that handles all the different
        // materials you have in the scenefile.
        // TODO: compare between directly shading the path segments and shading
        // path segments that have been reshuffled to be contiguous in memory.

        thrust::sort_by_key(thrust::device,
            dev_intersections,
            dev_intersections + num_paths,
            dev_paths, compareMatID());

        shadeFakeMaterial<<<numblocksPathSegmentTracing, blockSize1d>>>(
            iter,
            num_paths,
            dev_intersections,
            dev_paths,
            dev_materials,
            dev_textures,
            dev_pixels,
            dev_albedo,
            dev_normal
        );

        PathSegment* new_end = thrust::partition(thrust::device, dev_paths, dev_paths + num_paths, isActive());
        num_paths = new_end - dev_paths;

        if (num_paths <= 0) {
            iterationComplete = true;
        }

        if (depth > traceDepth)
        {
            iterationComplete = true;
        }

        if (guiData != NULL)
        {
            guiData->TracedDepth = depth;
        }
    }

    // Assemble this iteration and apply it to the image
    dim3 numBlocksPixels = (pixelcount + blockSize1d - 1) / blockSize1d;
    finalGather<<<numBlocksPixels, blockSize1d>>>(pixelcount, dev_image, dev_paths);

    // Apply denoising
    if ((iter == hst_scene->state.iterations || iter % DENOISE_INTERVAL == 0) && DENOISE) {
        denoiseImage();
        blendImage << <numBlocksPixels, blockSize1d >> > (dev_image, dev_output, pixelcount);
    }
    ///////////////////////////////////////////////////////////////////////////

    // Send results to OpenGL buffer for rendering
    sendImageToPBO<<<blocksPerGrid2d, blockSize2d>>>(pbo, cam.resolution, iter, dev_image);

    // Retrieve image from GPU
    hipMemcpy(hst_scene->state.image.data(), dev_image,
        pixelcount * sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("pathtrace");
}
